#include <algorithm>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <fstream>
#include <vector>
#include <chrono>
#include <string>
#include <thread>
#include "../include/Scanner.h"


void writeWeightData(std::string ofname, Scanner& scanner)
{
  // std::fstream ofs;
  // ofs.open(ofname, std::fstream::out | std::fstream::binary);
  // for(auto row_sum : scanner.row_sums) ofs.write(reinterpret_cast<char*>(&row_sum), sizeof(double));
  // for(auto col_sum : scanner.col_sums) ofs.write(reinterpret_cast<char*>(&col_sum), sizeof(double));
  // ofs.close();
}

void writeVector(std::string ofname, std::vector<double>& vec)
{
  // std::fstream ofs;
  // ofs.open(ofname, std::fstream::out | std::fstream::binary);
  // for(auto element : vec) ofs.write(reinterpret_cast<char*>(&element), sizeof(double));
  // ofs.close();
}

void printPoint(double point[], std::string prefix="", std::string suffix="")
{
  std::cout << prefix << "(" << point[0] << "," << point[1] << ")" << suffix;
}

void writePpmHeader(std::string ofname, int width, int height)
{
  std::fstream ofs;
  ofs.open(ofname, std::fstream::out | std::fstream::binary);
  ofs << "P2\n" << width << ' ' << height << "\n255\n";
  ofs.close();
}

void writePpmData(std::string ofname, double *data, int width, double max_val, double min_val)
{
  std::fstream ofs;
  ofs.open(ofname, std::fstream::out | std::fstream::binary | std::ios_base::app);
  for (int w = 0; w < width; ++w)
  {
    ofs << (int)(255.0*(data[w] - min_val)/(max_val - min_val)) << '\n';
  }
  ofs.close();
}

void readFile(std::string fname, double *vec, int size)
{
  std::fstream fs;
  fs.open(fname, std::fstream::in | std::fstream::binary);
  if (!fs.is_open())
  {
    std::cerr << "Can't find input file " << fname << "\n";
    exit(-1);
  }
  
  double buffer;
  for(int i=0; i<size; i++)
  {
    fs.read(reinterpret_cast<char*>(&buffer), sizeof(double));
    vec[i] = buffer;
  }
  fs.close();
}

void readWeightData(std::string fname, Scanner &scanner, double relax_param)
{
  // const double weight_cap = 1.0;
  // double buffer;
  // std::fstream fs;
  // fs.open(fname, std::fstream::in | std::fstream::binary);

  // if (!fs.is_open())
  // {
  //   std::cerr << "Can't find input file " << fname << "\n";
  //   exit(-1);
  // }

  // for(int i=0; i<scanner.num_pixels*scanner.num_pixels; i++)
  // {
  //   fs.read(reinterpret_cast<char*>(&buffer), sizeof(double));
  //   scanner.row_sums[i] = 1.0 / (buffer + 0.000001);
  // }

  // for(int i=0; i<scanner.num_views*scanner.num_detectors; i++)
  // {
  //   fs.read(reinterpret_cast<char*>(&buffer), sizeof(double));
  //   scanner.col_sums[i] = relax_param / (buffer + 0.000001);
  // }
  // fs.close();
}

void PrintProjectionParameters(Scanner scanner)
{
  std::cout << "scanning_radius: " << scanner.scanning_radius << '\n'
    << "detector_length: " << scanner.detector_length << '\n'
    << "num_pixels: " << scanner.num_pixels << "x" << scanner.num_pixels << '\n'
    << "num_views: " << scanner.num_views << '\n'
    << "num_detectors: " << scanner.num_detectors << '\n'
    << "phantom_radius: " << scanner.phantom_radius << '\n'
    << "field_of_view: " << scanner.field_of_view << std::endl;
}

int main(int argc, const char* argv[])
{
  int sysSize = (argc <= 1) ? 128 : std::atoi(argv[1]);
  double fov = (argc <= 2) ? 360.0 : (double)std::atof(argv[2]);
  char input_img = (argc <= 3) ? 'u' : *argv[3];
  char operation = (argc <= 4) ? 'p' : *argv[4];
  int sart_iter = (argc <= 5) ? 5 : std::atoi(argv[5]);
  double relax_param = (argc <= 6) ? 1.0 : (double)std::atof(argv[6]);

  std::string in_file_prefix = "input/unit_disc_";
  std::string out_file_prefix = "output/sino_unit_disc_";
  std::string img_out_file_prefix = "output/img_unit_disc_";
  std::string sart_out_file_prefix = "sart_output/sart_unit_disc_";
  std::string sart_weight_prefix = "sart_weights/sart_weight_";

  if(input_img == 'p')
  {
    in_file_prefix = "input/phantom_";
    out_file_prefix = "output/sino_phantom_";
    img_out_file_prefix = "output/img_phantom_";
    sart_out_file_prefix = "sart_output/sart_phantom_";
  }
  
  Scanner scanner = { 50.0, 40.0, sysSize, sysSize, sysSize, 10.0, fov };
  PrintProjectionParameters(scanner);
  int total_pixels = scanner.num_pixels*scanner.num_pixels;
  int total_detectors = scanner.num_detectors*scanner.num_views;
  
  double *img;
  double *sinogram;
  hipMallocManaged(&img, total_pixels*sizeof(double));
  hipMallocManaged(&sinogram, total_detectors*sizeof(double));

  readFile(in_file_prefix + std::to_string(scanner.num_pixels) + ".dat", img, total_pixels);
  sart_weight_prefix
    .append(std::to_string(scanner.num_pixels))
    .append("_")
    .append(std::to_string((int)scanner.field_of_view))
    .append("_")
    .append(std::to_string((int)scanner.detector_length))
    .append(".dat");

  // printSums(scanner);
  
////////////////////////
// Forward Projection
////////////////////////

  std::chrono::time_point<std::chrono::system_clock> start, end;
  std::chrono::duration<double> elapsed_seconds;

  start = std::chrono::system_clock::now();
  // project(scanner, img.data(), sinogram.data(), 0, scanner.num_views, ProjectionDirection::Forward);
  project<<<1, 1>>>(scanner, img, sinogram, 0, scanner.num_views, ProjectionDirection::Forward);
  hipDeviceSynchronize();
  end = std::chrono::system_clock::now(); 
  elapsed_seconds = end - start;
  std::cout << "elapsed time: " << elapsed_seconds.count() << "s\n";
  
  writePpmHeader(out_file_prefix + std::to_string(scanner.num_pixels) + ".ppm", scanner.num_detectors, scanner.num_views);
  double sino_max = *std::max_element(sinogram, sinogram + total_detectors);
  writePpmData(out_file_prefix + std::to_string(scanner.num_pixels) + ".ppm", sinogram, total_detectors, sino_max, 0.0);

////////////////////////
// Ramp Filtering
////////////////////////

//   std::chrono::time_point<std::chrono::system_clock> start, end;
//   std::chrono::duration<double> elapsed_seconds;

//   if(operation == 'f')
//   {
//     start = std::chrono::system_clock::now();
//     scanner.rampFilter(scanner.num_detectors, sinogram.data());
//     end = std::chrono::system_clock::now(); 
//     elapsed_seconds = end - start;
//     std::cout << "elapsed time: " << elapsed_seconds.count() << "s\n";
//   }
  
// ////////////////////////
// // Back Projection
// ////////////////////////

//   if((operation == 'f') || (operation == 'b'))
//   {
//     project(scanner, &img, &sinogram, ProjectionDirection::Backward);
    
//     writePpmHeader(img_out_file_prefix + std::to_string(scanner.num_pixels) + ".ppm", scanner.num_detectors, scanner.num_views);
//     double img_max = *std::max_element(img.begin(), img.end());
//     double img_min = *std::min_element(img.begin(), img.end());
//     writePpmData(img_out_file_prefix + std::to_string(scanner.num_pixels) + ".ppm", img, total_pixels, img_max, img_min);
//   }

// ////////////////////////
// // SART
// ////////////////////////
  
//   if(operation == 's')
//   {
//     readWeightData(sart_weight_prefix, scanner, relax_param);

//     std::vector<double> sinogram_error, img_error;
//     sinogram_error.resize(total_detectors);
//     img_error.resize(total_pixels);
//     std::string sart_out_fname;

//     std::fill(img.begin(), img.end(), 0.0); 
//     // project(scanner, &img, &sinogram, ProjectionDirection::Backward);

//     for(int i=0; i<sart_iter; i++)
//     {
//       std::cout << "\nSART Iter: " << i << "\n";
//       //compute error
//       project(scanner, &img, &sinogram_error, ProjectionDirection::Forward);
//       std::transform(sinogram_error.begin(), sinogram_error.end(), sinogram.begin(), sinogram_error.begin(), std::minus<double>());

//       //apply weights and project error
//       std::transform(sinogram_error.begin(), sinogram_error.end(), scanner.row_sums.begin(), sinogram_error.begin(), std::multiplies<double>()); 
//       project(scanner, &img_error, &sinogram_error, ProjectionDirection::Backward);
//       std::transform(img_error.begin(), img_error.end(), scanner.col_sums.begin(), img_error.begin(), std::multiplies<double>());

//       //update img
//       std::transform(img.begin(), img.end(), img_error.begin(), img.begin(), std::minus<double>());

//       sart_out_fname = sart_out_file_prefix;
//       sart_out_fname.append(std::to_string(scanner.num_pixels))
//         .append("_")
//         .append(std::to_string((int)scanner.field_of_view))
//         .append("_")
//         .append(std::to_string(i))
//         .append(".ppm");

//       writePpmHeader(sart_out_fname, scanner.num_pixels, scanner.num_pixels);
//       double _max = *std::max_element(img.begin(), img.end());
//       double _min = *std::min_element(img.begin(), img.end());
//       std::cout<< "img "  << _min << " - " << _max << "\n";
//       writePpmData(sart_out_fname, img, total_pixels, _max, _min);
//     }
  // }

  return 0;
}